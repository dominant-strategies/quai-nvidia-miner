#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <chrono>
#include <mutex>

#include "constants.h"
#include "uv.h"
#include "messages.h"
#include "blake3.cu"
#include "worker.h"
#include "template.h"
#include "mining.h"
#include "getopt.h"
#include "log.h"
#include<unistd.h>

std::atomic<uint32_t> found_solutions{0};

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::duration<double> duration_t;
typedef std::chrono::time_point<std::chrono::high_resolution_clock> time_point_t;

uv_loop_t *loop;
uv_stream_t *tcp;

time_point_t start_time = Time::now();

std::atomic<int> gpu_count;
std::atomic<int> worker_count;
std::atomic<uint64_t> total_mining_count;
std::atomic<uint64_t> device_mining_count[max_gpu_num];
bool use_device[max_gpu_num];

int port = 8008;
char broker_ip[16];
uv_timer_t reconnect_timer;
uv_tcp_t *uv_socket;
uv_connect_t *uv_connect;

void setup_gpu_worker_count(int _gpu_count, int _worker_count)
{
    gpu_count.store(_gpu_count);
    worker_count.store(_worker_count);
}

void on_write_end(uv_write_t *req, int status)
{
    if (status < 0)
    {
        LOGERR("error on_write_end %d\n", status);
    }
    free(req);
}

std::mutex write_mutex;
uint8_t write_buffer[4096 * 1024];
void submit_new_block(mining_worker_t *worker)
{
    expire_template_for_new_block(load_worker__template(worker));

    const std::lock_guard<std::mutex> lock(write_mutex);

    uint8_t temp_write_buffer[4096 * 1024];


    char method_str[] = "{\"method\":\"quai_rawHeader\",\"params\":[\"";      // Create the JSON-RPC to proxy including the necessary hashes
    ssize_t buf_size = write_new_block(worker, temp_write_buffer);

    char* ascii_string = (char*)malloc(buf_size*2 + 1);
    for (int i = 0; i < NONCE_LEN; i++) {
        sprintf(&ascii_string[i*2], "%02x", temp_write_buffer[i]);
    }
    ascii_string[buf_size*2] = '\0';

    char method_str2[] = "\"],\"id\":1,\"jsonrpc\":\"2.0\"}\n";

    memcpy(write_buffer, method_str, strlen(method_str));
    memcpy(write_buffer + strlen(method_str), ascii_string, strlen(ascii_string));
    memcpy(write_buffer + strlen(method_str) + strlen(ascii_string), method_str2, strlen(method_str2));

    uv_buf_t buf = uv_buf_init((char *)write_buffer, strlen(method_str) + strlen(ascii_string) + strlen(method_str2));
    print_hex("new solution: nonce", (uint8_t *) hasher_buf(worker, true), NONCE_LEN);
    print_hex("new solution: hash", (uint8_t *) hasher_hash(worker, true), 32);

    uv_write_t *write_req = (uv_write_t *)malloc(sizeof(uv_write_t));
    uint32_t buf_count = 1;

    uv_write(write_req, tcp, &buf, buf_count, on_write_end);
    free(ascii_string);
    LOG("Sent solution to proxy\n");
    found_solutions.fetch_add(1, std::memory_order_relaxed);
}

void mine_with_timer(uv_timer_t *timer);

static void register_proxy(uv_stream_t* tcp)
{
    char method_str[] = "{\"method\":\"quai_submitLogin\",\"params\":[\"0x0000000000000000000000000000000000000001\",\"password\"],\"id\":1,\"jsonrpc\":\"2.0\"}\n";

    uv_buf_t buf = uv_buf_init(method_str, strlen(method_str));

    uv_write_t* write_req = (uv_write_t *)malloc(sizeof(uv_write_t));
    write_req->data = method_str;

    uv_write(write_req, tcp, &buf, 1, on_write_end);
    
    LOG("Proxy registered\n");
}

void mine(mining_worker_t *worker)
{
    time_point_t start = Time::now();

    if (!ready_to_mine())
    {
        worker->timer.data = worker;
        uv_timer_start(&worker->timer, mine_with_timer, 500, 0);
    } else {
        mining_count.fetch_add(mining_steps);
        setup_template(worker, load_template(0));
        start_worker_mining(worker);

        // duration_t elapsed = Time::now() - start;
        // LOG("=== mining time: %fs\n", elapsed.count());
    }
}

void mine_with_req(uv_work_t *req)
{
    mining_worker_t *worker = load_req_worker(req);
    mine(worker);
}

void mine_with_async(uv_async_t *handle)
{
    mining_worker_t *worker = (mining_worker_t *)handle->data;
    mine(worker);
}

void mine_with_timer(uv_timer_t *timer)
{
    mining_worker_t *worker = (mining_worker_t *)timer->data;
    mine(worker);
}

void after_mine(uv_work_t *req, int status)
{
    return;
}

void worker_stream_callback(hipStream_t stream, hipError_t status, void *data)
{
    mining_worker_t *worker = (mining_worker_t *)data;
    if (hasher_found_good_hash(worker, true))
    {
        store_worker_found_good_hash(worker, true);
        submit_new_block(worker);
    }

    mining_template_t *template_ptr = load_worker__template(worker);
    uint32_t chain_index = 0;
    mining_count.fetch_sub(mining_steps);
    mining_count.fetch_add(hasher_hash_count(worker, true));
    total_mining_count.fetch_add(hasher_hash_count(worker, true));
    device_mining_count[worker->device_id].fetch_add(hasher_hash_count(worker, true));
    free_template(template_ptr);
    worker->async.data = worker;
    uv_async_send(&(worker->async));
}

void start_mining()
{
    assert(mining_templates_initialized == true);

    start_time = Time::now();

    for (uint32_t i = 0; i < worker_count.load(); i++)
    {
        if (use_device[mining_workers[i].device_id])
        {
            uv_queue_work(loop, &req[i], mine_with_req, after_mine);
        }
    }
}

void start_mining_if_needed()
{
    if (!mining_templates_initialized)
    {
        if (load_template(0) != NULL)
        {
            LOG("All templates initialized\n")
            mining_templates_initialized = true;
            start_mining();
        }
    }
}

void alloc_buffer(uv_handle_t *handle, size_t suggested_size, uv_buf_t *buf)
{
    buf->base = (char *)malloc(suggested_size);
    buf->len = suggested_size;
}

void log_hashrate(uv_timer_t *timer)
{
    time_point_t current_time = Time::now();
    if (current_time > start_time)
    {
        duration_t eplased = current_time - start_time;
        LOG("hashrate: %.0f MH/s ", total_mining_count.load() / eplased.count() / 1000000);
        for (int i = 0; i < gpu_count; i++)
        {
            LOG_WITHOUT_TS("gpu%d: %.0f MH/s ", i, device_mining_count[i].load() / eplased.count() / 1000000);
        }
        LOG_WITHOUT_TS("solutions: %u\n", found_solutions.load(std::memory_order_relaxed));
    }
}

uint8_t read_buf[2048 * 1024];
blob_t read_blob = {read_buf, 0};

server_message_t *decode_buf(const uv_buf_t *buf, ssize_t nread) {
    blob_t read_blob = * (blob_t*)malloc(sizeof(blob_t));
    read_blob.blob = (uint8_t*)malloc(nread * sizeof(uint8_t));
    read_blob.len = nread;

    memcpy(read_blob.blob, buf->base, nread);

    return decode_server_message(&read_blob);
}

void connect_to_broker();

void try_to_reconnect(uv_timer_t *timer){
    read_blob.len = 0;
    free(uv_socket);
    free(uv_connect);
    connect_to_broker();
    uv_timer_stop(timer);
}

void on_read(uv_stream_t *server, ssize_t nread, const uv_buf_t *buf)
{
    // LOG("Received %d bytes from server\n", nread);
    if (nread < 0)
    {
        LOGERR("error on_read %ld: might be that the full node is not synced, or miner wallets are not setup, try to reconnect\n", nread);
        uv_timer_start(&reconnect_timer, try_to_reconnect, 5000, 0);
        return;
    }

    if (nread == 0)
    {
        LOG("No data received\n");
        return;
    }

    LOG("Received new header from server\n");
    server_message_t* server_msg = decode_buf(buf, nread);

    if (server_msg) {
        switch (server_msg->kind)
        {
            case JOBS:
                update_templates(server_msg->job);
                start_mining_if_needed();
                break;
        }
        free_server_message_except_jobs(server_msg);
    }

    free(buf->base);
}

void on_connect(uv_connect_t *req, int status)
{
    if (status < 0)
    {
        LOGERR("connection error %d: might be that the full node is not reachable, try to reconnect\n", status);
        uv_timer_start(&reconnect_timer, try_to_reconnect, 1000, 0);
        return;
    }

    tcp = req->handle;
    register_proxy((uv_stream_t*)tcp);
    int result = uv_read_start(req->handle, alloc_buffer, on_read);
}

void connect_to_broker(){
    uv_socket = (uv_tcp_t *)malloc(sizeof(uv_tcp_t));
    uv_tcp_nodelay(uv_socket, 1);
    
    struct sockaddr_in dest;
    uv_ip4_addr(broker_ip, port, &dest);
    
    uv_tcp_init(loop, uv_socket);

    uv_tcp_bind(uv_socket, (struct sockaddr *)&dest, 0);

    uv_connect = (uv_connect_t *)malloc(sizeof(uv_connect_t));

    uv_tcp_connect(uv_connect, uv_socket, (const struct sockaddr *)&dest, on_connect);

}

bool is_valid_ip_address(char *ip_address)
{
    struct sockaddr_in sa;
    int result = inet_pton(AF_INET, ip_address, &(sa.sin_addr));
    return result != 0;
}

int hostname_to_ip(char *ip_address, char *hostname)
{
    struct addrinfo hints, *servinfo;
    memset(&hints, 0, sizeof hints);
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_STREAM;

    int res = getaddrinfo(hostname, NULL, &hints, &servinfo);
    if (res != 0)
    {
        LOGERR("getaddrinfo: %s\n", gai_strerror(res));
        return 1;
    }

    struct sockaddr_in *h = (struct sockaddr_in *)servinfo->ai_addr;
    strcpy(ip_address, inet_ntoa(h->sin_addr));

    freeaddrinfo(servinfo);
    return 0;
}
#ifndef MINER_VERSION
#define MINER_VERSION "unknown"
#endif

int main(int argc, char **argv)
{
    setbuf(stdout, NULL);

    #ifdef _WIN32
    WSADATA wsa;
    // current winsocket version is 2.2
    int rc = WSAStartup(MAKEWORD(2, 2), &wsa);
    if (rc != 0)
    {
        LOGERR("Initialize winsock failed: %d\n", rc);
        exit(1);
    }
    #endif

    LOG("Running gpu-miner version : %s\n", MINER_VERSION);

    int gpu_count = 0;
    hipGetDeviceCount(&gpu_count);
    LOG("GPU count: %d\n", gpu_count);
    for (int i = 0; i < gpu_count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        LOG("GPU #%d - %s has #%d cores\n", i, prop.name, get_device_cores(i));
        use_device[i] = true;
    }

    int command;
    while ((command = getopt(argc, argv, "p:g:a:")) != -1)
    {
        switch (command)
        {
        case 'p':
            port = atoi(optarg);
            break;
        case 'a':
            if (is_valid_ip_address(optarg))
            {
                strcpy(broker_ip, optarg);
            }
            else
            {
                hostname_to_ip(broker_ip, optarg);
            }
            break;

        case 'g':
            for (int i = 0; i < gpu_count; i++)
            {
                use_device[i] = false;
            }
            optind--;
            for (; optind < argc && *argv[optind] != '-'; optind++)
            {
                int device = atoi(argv[optind]);
                if (device < 0 || device >= gpu_count) {
                    LOGERR("Invalid gpu index %d\n", device);
                    exit(1);
                }
                use_device[device] = true;
            }
            break;
        default:
            LOGERR("Invalid command %c\n", command);
            exit(1);
        }
    }
    LOG("will connect to broker @%s:%d\n", broker_ip, port);

    #ifdef __linux__
    signal(SIGPIPE, SIG_IGN);
    #endif

    mining_workers_init(gpu_count);
    LOG("worker count: %d\n", gpu_count);
    setup_gpu_worker_count(gpu_count, gpu_count * parallel_mining_works_per_gpu);

    loop = uv_default_loop();
    uv_timer_init(loop, &reconnect_timer);
    connect_to_broker();

    for (int i = 0; i < worker_count; i++)
    {
        uv_async_init(loop, &(mining_workers[i].async), mine_with_async);
        uv_timer_init(loop, &(mining_workers[i].timer));
    }

    uv_timer_t log_timer;
    uv_timer_init(loop, &log_timer);
    uv_timer_start(&log_timer, log_hashrate, 5000, 20000);

    uv_run(loop, UV_RUN_DEFAULT);

    uv_loop_close(loop);
    free(loop);

    return 0;
}
